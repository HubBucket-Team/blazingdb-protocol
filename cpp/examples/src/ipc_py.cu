#include <iostream>

#include <blazingdb/protocol/api.h>
#include <hip/hip_runtime.h>
#include "gdf/GDFColumn.cuh"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


static void* _CudaIpcMemHandlerFrom (const uint8_t *bytes) {
  void * response = nullptr;
  hipIpcMemHandle_t ipc_memhandle;

  memcpy((int8_t*)&ipc_memhandle, bytes, sizeof(ipc_memhandle));
  hipIpcOpenMemHandle((void **)&response, ipc_memhandle, hipIpcMemLazyEnablePeerAccess);
  cudaCheckErrors("IPC handle fail");

  return response;
}

int main() {
   blazingdb::protocol::UnixSocketConnection connection(
       {"/tmp/demo.socket", std::allocator<char>()});
   
   blazingdb::protocol::Server server(connection);
   server.handle([](const blazingdb::protocol::Buffer &buffer)
                     -> blazingdb::protocol::Buffer {
     
     std::cout << buffer.data() << std::endl;
    
    void *pointer = _CudaIpcMemHandlerFrom(buffer.data());
    libgdf::gdf_column column {
        .data = pointer,                       /**< Pointer to the columns data */
        .valid = (unsigned char *)pointer,            /**< Pointer to the columns validity bit mask where the 'i'th bit indicates if the 'i'th row is NULL */
        .size = 32,               /**< Number of data elements in the columns data buffer*/
        .dtype = (libgdf::gdf_dtype)1,                  /**< The datatype of the column's data */
        .null_count = 0,         /**< The number of NULL values in the column's data */
        .dtype_info = libgdf::gdf_dtype_extra_info{
            .time_unit = (libgdf::gdf_time_unit)0,
        }
    };
    libgdf::print_column(&column);
    return blazingdb::protocol::Buffer(
         reinterpret_cast<const std::uint8_t *>("BlazingDB Response"), 18);
   });

  return 0;
}
